#include "cuda/fft.cuh"
#include <hip/hip_runtime.h>

#include <stdio.h>

float time_cuda(int N);
void print_cuda(int N);

int main(int argc, char **argv) {

	printf("N  | time (ms)\n");

	for (int i = 2; i <= 15; i++)
	{
		printf("%d %f\n", 1 << i, time_cuda(1 << i));
	}

	return 0;
}

float time_cuda(int N)
{
	double *h_in = new double[2*N];

	for (int i = 0; i < 2*N; i++) {
		h_in[i] = (!i) ? 1 : 0; // delta function
	}

	fft::cuda::Complex *d_in, *d_out;
	hipMalloc(&d_in, N * sizeof(fft::cuda::Complex));
	hipMalloc(&d_out, N * sizeof(fft::cuda::Complex));
	hipMemcpy(d_in, h_in, N * sizeof(fft::cuda::Complex), hipMemcpyHostToDevice);

	fft::cuda::fftPlan plan(N);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	fft::cuda::FFT(d_in, d_out, plan, true);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	delete[] h_in;
	hipFree(d_in);
	hipFree(d_out);

	return milliseconds;
}
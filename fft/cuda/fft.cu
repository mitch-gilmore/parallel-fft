#include "hip/hip_runtime.h"
#include "fft.cuh"

namespace fft
{
	namespace cuda
	{	
		namespace kernels
		{
			using fft::cuda::Complex;
			using fft::cuda::I;
			using fft::cuda::fftPlan;
			
			__global__ static void FFT(Complex *Fout, Complex *Fin, Complex *omega, int *indeces, unsigned int N, unsigned int stages, bool forward)
			{
				int idx = threadIdx.x + blockIdx.x * blockDim.x;
				
				Complex twiddle = (forward) ? omega[N / 2] : hipConj(omega[N / 2]);
				
				if (idx < N/2)
				{
					// stockhams shuffle
					{
						int Bsize = 2;
						int Nblocks = (int)N / Bsize;
						
						int nb = idx / Bsize;
						int pairs = idx % Bsize / 2;
						int k = pairs + nb * Bsize;
						
						Complex phase = (forward) ? omega[Nblocks * pairs] : hipConj(omega[Nblocks * pairs]);
						Complex low = Fin[k];
						Complex high = hipCmul(phase, Fin[k + Bsize / 2]);
						
						Fout[indeces[k]] = hipCadd(low, high);
						Fout[indeces[k + Bsize / 2]] = hipCadd(low, hipCmul(twiddle, high));
						
						__syncthreads();
					}
					
					for (int s = 1; s < stages; s++)
					{
						int Bsize = pow(2, s + 1);
						int Nblocks = N / Bsize;
						
						int nb = idx / Bsize;
						int pairs = idx % Bsize / 2;
						int k = pairs + nb * Bsize;
						
						Complex phase = (forward) ? omega[Nblocks * pairs] : hipConj(omega[Nblocks * pairs]);
						Complex low = Fout[k];
						Complex high = hipCmul(phase, Fout[k + Bsize / 2]);
						Fout[k] = hipCadd(low, high);
						Fout[k + Bsize / 2] = hipCadd(low, hipCmul(twiddle, high));
						
						__syncthreads();
					}
				}
			}
			
			__forceinline__ __device__ static unsigned int reverseBits(unsigned int x, unsigned int stages, unsigned int N)
			{
				unsigned int xrev = 0;
				// unsigned int p = log2(N); // p = 4
				unsigned int n;
				unsigned int power = N;
				
				for (unsigned int i = 0; i < stages; i++)
				{
					n = x % 2; // find lowest bit
					power /= 2;
					xrev += n * power; //  add to highest 2^3
					x /= 2;
				}
				
				return xrev;
			}
			
			__global__ static void makeIndexShuffle(int *indeces, unsigned int stages, unsigned int N)
			{
				int idx = threadIdx.x + blockIdx.x * blockDim.x;
				if (idx < N)
					indeces[idx] = reverseBits(idx, stages, N);
			}
			
			__global__ void static makePhase(Complex *omega, unsigned int N)
			{
				int idx = threadIdx.x + blockIdx.x * blockDim.x;
				
				if (idx < N)
				{
					double phase = -1.0 * 2.0 * M_PI * (double)idx / (double)N;
					omega[idx].x = cos(phase);
					omega[idx].y = sin(phase);
				}
			}
		}
		
		// Constructor
		fftPlan::fftPlan(unsigned int N)
		{
			this->N = N;
			this->stages = log2(N);
			
			hipMalloc((void**)&this->omega, N * sizeof(Complex));
			this->makePhase();
			
			hipMalloc((void**)&this->indexShuffle, N * sizeof(int));
			this->makeIndexShuffle();
		}
		
		// Destructor
		fftPlan::~fftPlan()
		{
			hipFree(this->omega);
			hipFree(this->indexShuffle);
		}
		
		// Computes all the complex roots of unity
		void fftPlan::makePhase()
		{
			int threadsPerBlock = 32;
			int numBlocks = (this->N + threadsPerBlock - 1) / threadsPerBlock;

			kernels::makePhase<<<numBlocks, threadsPerBlock>>>(this->omega, this->N);
		}

		// Computes the index shuffle
		void fftPlan::makeIndexShuffle()
		{
			int threadsPerBlock = 32;
			int numBlocks = (this->N + threadsPerBlock - 1) / threadsPerBlock;

			kernels::makeIndexShuffle<<<numBlocks, threadsPerBlock>>>(this->indexShuffle, this->stages, this->N);
		}
		
		void FFT(Complex *Fout, Complex *Fin, fftPlan plan, bool forward)
		{
			
			int threadsPerBlock = 32;
			int numBlocks = (plan.N + threadsPerBlock - 1) / (threadsPerBlock * 2);
			
			kernels::FFT<<<numBlocks, threadsPerBlock>>>(Fout, Fin, plan.omega, plan.indexShuffle, plan.N, plan.stages, forward);
		}
	}
}